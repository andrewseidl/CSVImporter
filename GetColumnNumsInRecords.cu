#include "hip/hip_runtime.h"
/******************************************************************************
* Copyright (c) 2016-2018, Brian Kennedy.  All rights reserved.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*
******************************************************************************/

/******************************************************************************
*
* See https://github.com/Simantex/CSVImporter for repository and documentation.
*
******************************************************************************/
#include "CSV_kernel_declarations.cuh"
#include "csvImporter.h"

extern "C" void
launch_GetColumnNumsInRecords(uint32_t *  d_ScanRecs, uint32_t *  d_ScanCols,
uint32_t *  d_OrdinalsRecsToCols, uint16_t * d_ColNumInRecs, uint32_t ValuesCount)
{

	// Call stream compact kernel.
	int iThreads = 256;
	float fBlocks = (float)ValuesCount / ((float)iThreads);
	int iBlocks = ValuesCount / iThreads;
	fBlocks = fBlocks - iBlocks;
	if (fBlocks > 0)
		iBlocks++;

	GetColumnNumsInRecords <<< iBlocks, iThreads >>>(d_ScanRecs, d_ScanCols, d_OrdinalsRecsToCols, d_ColNumInRecs, ValuesCount);

	Check_cuda_Errors("GetColumnNumsInRecords");
}

__global__ void GetColumnNumsInRecords(uint32_t *  d_ScanRecs, uint32_t *  d_ScanCols,
	uint32_t *  d_OrdinalsRecsToCols, uint16_t * d_ColNumInRecs, uint32_t ValuesCount )
{
	int ix = blockIdx.x * blockDim.x + threadIdx.x;

	if (ix >= ValuesCount) return;  // ignore anything in last block beyond source arrays length.

	uint32_t firstcol;
	uint32_t rec;
	uint32_t col;
	uint32_t colinrec;
	if (ix == 0)
	{
		rec = 0;
		firstcol = 0;
		col = 0;
	}
	else
	{
		rec = d_ScanRecs[ix];
		col = d_ScanCols[ix];
	}
	if (rec == 0) firstcol = 0;
	else firstcol = d_OrdinalsRecsToCols[rec - 1] + 1;
	colinrec = col - firstcol;
	d_ColNumInRecs[ix] = (uint16_t)colinrec;

	return;
}
