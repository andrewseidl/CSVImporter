#include "hip/hip_runtime.h"
﻿/******************************************************************************
* Copyright (c) 2016-2018, Brian Kennedy.  All rights reserved.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*
******************************************************************************/

/******************************************************************************
*
* See https://github.com/Simantex/CSVImporter for repository and documentation.
*
******************************************************************************/

#include <stdio.h>
#include <string.h>
#include <errno.h>
#include <ctype.h>
#include <stdlib.h>
#include <iostream>
#include <sstream>
#include <stdint.h>
#include <time.h>

using namespace std;

// CUDA runtime
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include ""

// CUDA atomics
#include "sm_35_atomic_functions.h"

#include "CommonDefinitions.h"

#include "include/util/mgpucontext.h"
#include "include/kernels/scan.cuh"

#include <algorithm>

#include "csvImporter.h"
#include "CSV_kernel_declarations.cuh"

using namespace mgpu;

// function copies only col names on non-omitted columns.
extern "C" bool CopyOnlyUsedColNames(char * columnnames[], bool skucolflagsin[], uint16_t totalcolcount, uint16_t definedcolcount, int16_t * arrayUTF8charwidths, char ** usedcolumnnames[], bool * skucolflagsout[], uint16_t * usedcolcount)
{
	// can't define more cols than you have.
	if (definedcolcount > totalcolcount) return false;

	// count how many pointers we need.
	int cidx = 0;
	for (int idx = 0; idx < definedcolcount; idx++)
	{
		if (arrayUTF8charwidths[idx] > 0)
		{
			cidx++;
		}
	}
	*usedcolcount = cidx;

	// malloc the arrays.
	char ** tnamearray = new char*[cidx];
	bool * skucols = new bool[cidx];

	cidx = 0;
	for (int idx = 0; idx < definedcolcount; idx++)
	{

		if (arrayUTF8charwidths[idx] > 0)
		{
			tnamearray[cidx] = columnnames[idx];  // simply copy over the pointer.  don't reallocate.
			skucols[cidx] = skucolflagsin[idx];
			cidx++;
		}
	}
	*skucolflagsout = skucols;
	*usedcolumnnames = tnamearray;

	return true;
}

// Reads in first record of CSV file to get the column names and total column count.
extern "C" bool PreProcessFileHeaderRecord(char * filepath, char ** columnnames[], uint16_t * colcount, int64_t * seekafterheader, char delim)
{
	pCsvFileIn = fopen(filepath, "rb");
	if (pCsvFileIn == NULL) return false;

	fseek(pCsvFileIn, (int64_t)0, SEEK_END);
	CsvFileLength = ftell(pCsvFileIn);  // get length of file.
	fseek(pCsvFileIn, (int64_t)0, SEEK_SET);  // reset file ptr to beginning for read

	uint64_t readlen = 32768;  // assume this is the biggest record.
	char * tbuf = new char[readlen];
	if (CsvFileLength < readlen) readlen = CsvFileLength;
	fread(tbuf, sizeof(char), readlen, pCsvFileIn);  // read in the data.


	// alloc temp array for up to 1000 cols.
	char ** tnamearray = new char*[1000];


	int colindex = 0;  // count the columns.
	int colstartpos = 0;  // start of current col.
	int pastrecord = 0;
	bool recordfound = false;
	// now parse the data to get first record.
	for (int i = 0; i < (int)readlen; i++)
	{
		// if hit a line feed, this is the end of the record.
		if (tbuf[i] == '\n')
		{
			pastrecord = i + 1;  // position past this first record is next byte after linefeed.
			*colcount = colindex;  // count == index since index should be bumped past last record.
			recordfound = true;
			break;
		}
		// if find a delimiter or a CR, have completed a column.
		if ((tbuf[i] == delim) || (tbuf[i] == '\r'))
		{
			int colnamelen = i - colstartpos;  // get the length of the column name.
			// for a blank name create a name.
			if (colnamelen == 0)
			{
				tnamearray[colindex] = new char[14];
				sprintf(tnamearray[colindex], "##COLUMN %04d", colindex);
			}
			else
			{
				// here we could add a trim string.

				// for now just copy over the characters.
				tnamearray[colindex] = new char[colnamelen + 1];
				strncpy(tnamearray[colindex], tbuf + colstartpos, colnamelen);
				tnamearray[colindex][colnamelen] = 0;
			}
			colindex++;
			colstartpos = i + 1;  // past the delimiter.
		}
	}

	// if read all the way with no record found, return false.
	if (!recordfound) return false;

	// now allocate array to return and copy pointers over from temp array.
	char ** finalcolumnnames = new char*[colindex];  // alloc for number of name pointers.
	for (int i = 0; i < colindex; i++)
	{
		finalcolumnnames[i] = tnamearray[i];
	}
	*columnnames = finalcolumnnames;  // copy new array to pointer.

	*seekafterheader = (int64_t)pastrecord;

	fseek(pCsvFileIn, (int64_t)0, SEEK_SET);  // reset file ptr to beginning for read.

	// FOR NOW CLOSE THE FILE.
	fclose(pCsvFileIn);

	delete tnamearray;
	delete tbuf;

	return true;
}

// Deletes ALL column names (shared by both columnnames and usedcolumnnames) as well as deleting
// and 0-ing the columnnames and usedcolumnnames.
extern "C" bool DeleteFileHeaderNames(char ** columnnames[], uint16_t colcount, char ** usedcolumnnames[], bool * usedskuflags[])
{
	char ** colnames = *columnnames;

	for (int i = 0; i < colcount; i++)
	{
		delete colnames[i];
	}
	delete colnames;
	*columnnames = 0;

	colnames = *usedcolumnnames;
	delete colnames;
	*usedcolumnnames = 0;

	delete *usedskuflags;
	*usedskuflags = 0;

	return true;
}



// This functions is like the prior one, except its purpose is simply to return an initial SEEK point after the first record.
extern "C" bool SkipFileHeaderRecord(char * filepath, int64_t * seekafterheader)
{
	pCsvFileIn = fopen(filepath, "rb");
	if (pCsvFileIn == NULL) return false;

	fseek(pCsvFileIn, (int64_t)0, SEEK_END);
	CsvFileLength = ftell(pCsvFileIn);  // get length of file.
	fseek(pCsvFileIn, (int64_t)0, SEEK_SET);  // reset file ptr to beginning for read

	uint64_t readlen = 32768;  // assume this is the biggest record.
	char * tbuf = new char[readlen];
	if (CsvFileLength < readlen) readlen = CsvFileLength;
	fread(tbuf, sizeof(char), readlen, pCsvFileIn);  // read in the data.

	int pastrecord = 0;
	bool recordfound = false;
	// now parse the data to get first record.
	for (int i = 0; i < (int)readlen; i++)
	{
		// if hit a line feed, this is the end of the record.
		if (tbuf[i] == '\n')
		{
			pastrecord = i + 1;  // position past this first record is next byte after linefeed.
			recordfound = true;
			break;
		}
	}

	// if read all the way with no record found, return false.
	if (!recordfound) return false;

	*seekafterheader = (int64_t)pastrecord;

	fseek(pCsvFileIn, (int64_t)0, SEEK_SET);  // reset file ptr to beginning for read.

	// FOR NOW CLOSE THE FILE.
	fclose(pCsvFileIn);

	delete tbuf;

	return true;
}



// main function
//
// here we read the CSV file in and build the output arrays.
// this is set up for the older vertical bar delimiter or true comma-delimited files.
// pass in:
// the full path of the input CSV file
// an array of character widths per field  (this will determine how many characters to allow)
// the number of columns or fields
// deliminator character to look for.
// GPUResidentFlag = false means copy final data arrays to CPU, otherwise copy final data array to GPU
// Device or Host arrays of final column data,
// a multiplier for how many bytes to allocate per char.  normally 1 for ASCII, 3 for UTF-8.
// a byte alignment for the output arrays.  normally 8 (middleware standard), but 4 works here.
//
// NOTE: This function can be used as a more generic CSV importer into GPU arrays.
// Its use is anticipated for future import jobs.
// As part of Middleware for B2Bx, it will normally be set up for ASCII, 1 byte per char,
// 8-byte alignment, and use only 2 columns (for the Division and the SKU).
// Subsequent functions are specially tailored for Middleware.
//
// The purpose of the domiddleware flag is to limit the final copy to 2 columns, DIV and SKU.
// We also pass in the SKU and DIV col #s (defaults 0 and 1) in the INPUT file.
// The middleware outputs from this function will always put SKU in col 0 and DIV in col 1.
//
uint64_t importer_varcols(CudaContext& context, char * filepath,
	int16_t * arrayUTF8charwidths, uint16_t numdefinedcolumns, uint16_t numtotalcolumns,
	char delimiter, bool GPUResidentFlag, unsigned char ** dataColumnPtrs, unsigned int * dataColumnOffsets, int64_t initialseek = 0, uint8_t charmultiplier = 1, uint8_t bytesalignment = 8)
{
	// BELOW simply opens the file, calculates length and chunk sizes
	// returns 0 if no issues, 1 for no records, -1 for file error.
	int chunkret = CSVfilechunking(filepath);
	if (chunkret == -1)
	{
		printf("Error opening file %s.\r\n", filepath);
		return 0;
	}
	else if (chunkret == 1)
	{
		return 0;  // for no records.
	}
	// now just in case we have a header that is skipped, make sure we are 6 bytes are more.
	if ((CsvFileLength - initialseek) <= MAXCHAR0RECORDS)
	{
		return 0;   // for no records.
	}

	// NOW Initialize the buffers that will be reused in each chunk.
	// CPU versions (some for debugging only).
	InitializeCPUElements_REUSABLES(SufficientBytes);

	h_fieldUTF8charsizes = 0;
	checkCudaErrors(hipHostMalloc((void **)&h_fieldUTF8charsizes, numtotalcolumns * sizeof(uint16_t)));


	// here we simply alloc the array of pointers (not the data).
	checkCudaErrors(hipHostMalloc((void **)&h_fieldptrs, numtotalcolumns * sizeof(unsigned char *)));
	checkCudaErrors(hipHostMalloc((void **)&h_d_fieldptrs, numtotalcolumns * sizeof(unsigned char *)));
	// since these are arrays of pointers, 0 out the pointers.
	for (int xi = 0; xi < numtotalcolumns; xi++)
	{
		h_fieldptrs[xi] = (unsigned char *)0;
		h_d_fieldptrs[xi] = (unsigned char *)0;
	}
	printf("HOST ARRAY CARRIAGES ALLOC'ED AT (host) %llx (device) %llx.\r\n", (int64_t)h_fieldptrs, (int64_t)h_d_fieldptrs);

	int savedfieldcount = 0;  // this tracks the fields that are not discarded.

	// loop through the char sizes array.
	// two objectives:
	// 1. define byte sizes based on char multiplier and alignment.
	// 2. flesh out array for undefined columns at ends of records.
	for (int num = 0; num < numtotalcolumns; num++)
	{
		// if have gone past defined columns we make an ignore column.
		if (num >= numdefinedcolumns)
		{
			h_fieldUTF8charsizes[num] = -1;
		}
		// otherwise copy it over to the new fullsized array.
		else
		{
			// copy over the width as passed in, may not be an aligned value.
			// however, determines the max num of UTF-8 chars.
			h_fieldUTF8charsizes[num] = arrayUTF8charwidths[num];
		}

		// Also -1 is special case, meaning ignore column.  so don't try to align.
		if (h_fieldUTF8charsizes[num] == -1)
		{
			G_h_fieldbytewidths[num] = 0;  // 0 means ignore column.
		}
		else
		{
			// ENFORCE byte alignment for the byte width of the field.
			uint16_t tsiz = arrayUTF8charwidths[num] * charmultiplier;  // multiply by storage multiplier (e.g., 3 for UTF-8, 1 for ASCII)
			uint16_t tlowbits = tsiz & (uint16_t)(bytesalignment - 1);
			if (tlowbits != (uint16_t)0)
			{
				tsiz += ((uint16_t)bytesalignment - tlowbits);
			}
			G_h_fieldbytewidths[num] = tsiz;

			// bump the final field count.
			savedfieldcount++;
		}
	}

	// NOW Initialize the buffers that will be reused in each chunk.
	// Set length to max byte size in chunks plus 1 to pick up final value in exclusive scan.
	// NOTE: no need to do any more since should be ample and also rounded up to 256 byte boundary.
	InitializeGPUElements_REUSABLES(SufficientBytes);

	startseek = initialseek;  // just to make sure starting at the beginning or just after header.
	bool lastchunk = false;
	uint64_t bytestoread;
	uint32_t chunkbytes;
	for (int chunknum = 0; (chunknum < inumchunks) && (!lastchunk); chunknum++)
	{
		// print out 1-based:
		printf("Processing Chunk %d of %d%s.\r\n", chunknum + 1, inumchunks, (lastchunk ? " (last)" : ""));

		// here we clean up the full reusable GPU mem.
		MemsetGPUElements_REUSABLES(SufficientBytes);

		uint64_t testchunksize = SufficientBytes - OVERREAD;  // take off the overread to get the test chunk size.
		if ((CsvFileLength - startseek) <= testchunksize)
		{
			lastchunk = true;
			bytestoread = CsvFileLength - (uint64_t)startseek;
			chunkbytes = (uint32_t)bytestoread;  // this is the exact chunk size.
		}
		else
		{
			bytestoread = testchunksize;
		}

		fseek(pCsvFileIn, startseek, SEEK_SET);  // set file ptr to start for read
		fread(h_CsvBuffer_a, sizeof(char), bytestoread, pCsvFileIn);  // read in the chunk.

		// if not at last chunk, back up to a record terminator (linefeed).
		if (!lastchunk)
		{
			for (int bidx = (int)(bytestoread - 1); bidx >= 0; bidx--)
			{
				if (h_CsvBuffer_a[bidx] == 0x0a)
				{
					chunkbytes = bidx + 1;
					h_CsvBuffer_a[chunkbytes] = 0;  // make sure to 0 out next byte since that will copy.
					break;
				}
			}
		}
		// else if this is lastchunk, must close the file.
		else
		{
			fclose(pCsvFileIn);
		}

		if (chunkbytes == 0)
		{
			printf("Error: record break not found.\r\n");
			exit(0);
		}
		startseek += (uint64_t)chunkbytes;   // for next time.

		/////////////////////
		// Sometimes files are missing the final CR LF we use to identify a record.
		// In this case we patch on a final CR LF.
		// The host buffer has ample memory allocated to do so.
		if (lastchunk && (h_CsvBuffer_a[bytestoread - 1] != 0xa))
		{
			h_CsvBuffer_a[bytestoread] = 0xd;
			h_CsvBuffer_a[bytestoread + 1] = 0xa;
			chunkbytes += 2;
		}
		/////////////////////

		// START PROCESSING OF CHUNK.
		uint32_t chunkbytesplus1 = chunkbytes + 1;
		uint32_t chunkbytesplus1uint32 = (chunkbytes + 1) * sizeof(uint32_t);

		checkCudaErrors(hipMemcpy((void*)d_CsvBuffer, (void*)h_CsvBuffer_a, (size_t)chunkbytesplus1, hipMemcpyHostToDevice));

		// handle true comma delim inputs.
		// idea for now is to "preprocess" the file, then clean it up to look more like the
		// older vertical bar delim, except we will substitute (temporarily) a 0 for the vert bar
		// col delim.  the rest of the code will proceed more or less as before.

		launch_MarkCommas(d_CsvBuffer, d_QuoteBoundaryHeaders, d_CommaHeaders, d_RecordHeaders, d_ColumnHeaders, d_printingchars_flags, chunkbytes, delimiter);

		launch_DoubleQuotes(d_CsvBuffer, d_printingchars_flags, d_secondquotes, chunkbytes);

		launch_Merge2ndQuotesAndNonprinting(d_printingchars_flags, d_secondquotes, chunkbytes);

		printf("Starting Scan Record Headers (Commas).\n");
		Scan<MgpuScanTypeExc>(d_RecordHeaders, chunkbytesplus1,
			(uint32_t)0, mgpu::plus<uint32_t>(), (uint32_t *)0, (uint32_t*)0, d_RecordHeadersSCAN, context);

		// get records count.
		uint32_t recordstablecount_commas;
		checkCudaErrors(hipMemcpy(&recordstablecount_commas, d_RecordHeadersSCAN + chunkbytes, sizeof(uint32_t), hipMemcpyDeviceToHost));

		printf("Starting Scan Quote Boundaries (Commas).\n");
		Scan<MgpuScanTypeExc>(d_QuoteBoundaryHeaders, chunkbytesplus1,
			(uint32_t)0, mgpu::plus<uint32_t>(), (uint32_t *)0, (uint32_t*)0, d_QuoteBoundaryHeaders_SCAN, context);

		// get quote boundaries count.
		uint32_t quoteboundariescount;
		checkCudaErrors(hipMemcpy(&quoteboundariescount, d_QuoteBoundaryHeaders_SCAN + chunkbytes, sizeof(uint32_t), hipMemcpyDeviceToHost));

		// Records table is a table of record header locations.
		// Columns table is a table of PROSPECTIVE column header locations.
		// RecordsToColumns table is a table that maps record headers to locations in the Columns table.

		// do the prelim processing to get segmented scan.
		launch_RecordsProspectiveColumns_StreamCompact(d_RecordHeaders, d_QuoteBoundaryHeaders, d_RecordHeadersSCAN, d_QuoteBoundaryHeaders_SCAN, d_RecordsToQuoteBoundariesTable, chunkbytesplus1);

		// use an existing kernel here.
		launch_GetColumnNumsInRecords(d_RecordHeadersSCAN, d_QuoteBoundaryHeaders_SCAN, d_RecordsToQuoteBoundariesTable, d_ColumnNumInRecord, chunkbytes);
		launch_FixColumnHeaderCommas(d_ColumnNumInRecord, d_CommaHeaders, d_ColumnHeaders, chunkbytes);

		// now we need to stream compact buffer, rec headers, and col headers for use later.
		printf("Starting Scan Printing Chars (Commas).\n");
		Scan<MgpuScanTypeExc>(d_printingchars_flags, chunkbytesplus1,
			(uint32_t)0, mgpu::plus<uint32_t>(), (uint32_t *)0, (uint32_t*)0, d_printingchars_SCAN, context);

		// get the final count of printing chars.
		uint32_t printingcharscount;
		checkCudaErrors(hipMemcpy(&printingcharscount, d_printingchars_SCAN + chunkbytes, sizeof(uint32_t), hipMemcpyDeviceToHost));

		// now reallocate buffer, record headers, and col headers.


		////////////***
		// For the replacement buffer must also add some space at the end for the CSV writer to read ahead a full chunk without a memory error.
		// This should only require 128 bytes.  Round up as a sanity check.
		uint32_t printingcharstemp = printingcharscount + 1 + 256;
		uint32_t mod = printingcharstemp % 256;
		if (mod > 0) printingcharstemp += (256 - mod);



		launch_BufferPrinting_StreamCompact(d_printingchars_flags, d_printingchars_SCAN,
			d_CsvBuffer, d_CsvBuffer_printing,
			d_RecordHeaders, d_RecordHeaders_printing,
			d_ColumnHeaders, d_ColumnHeaders_printing, chunkbytes);

		// now reintegrate into prior vertical bar col sep version.
		// essentially shorten up what we normally have.
		// shorten chunkbytes as needed.
		chunkbytes = printingcharscount;
		chunkbytesplus1 = chunkbytes + 1;
		chunkbytesplus1uint32 = chunkbytesplus1 * sizeof(uint32_t);

		launch_BuildCharsHeadersOnly(d_CsvBuffer_printing, d_RecordHeaders_printing, d_ColumnHeaders_printing, d_UTF8Headers, chunkbytesplus1);

		printf("Starting Scan UTF8 Headers.\n");

		// exclusive scan the ends headers so all positions for each zip will have same scan value.
		Scan<MgpuScanTypeExc>(d_UTF8Headers, chunkbytesplus1,
			(uint32_t)0, mgpu::plus<uint32_t>(), (uint32_t *)0, (uint32_t*)0, d_UTF8HeadersSCAN, context);


		// retrieve last value, the one past the end of the actual values.
		uint32_t * h_charstablecount = (uint32_t *)malloc(sizeof(uint32_t));

		// want the record at the end of the scan.  the pointer math adds CsvFileLength as uint32_t so equals a CsvFileLength * 4 bytes adjustment
		cudaStatus = hipMemcpy((void*)h_charstablecount, (void*)(d_UTF8HeadersSCAN + chunkbytes), 4, hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed (utf8 chars scan last)! %s", hipGetErrorString(cudaStatus));
			return 0;
		}
		uint32_t charstablecount = *h_charstablecount;

		free(h_charstablecount);

		printf("Starting Scan Record Headers.\n");
		Scan<MgpuScanTypeExc>(d_RecordHeaders_printing, chunkbytesplus1,
			(uint32_t)0, mgpu::plus<uint32_t>(), (uint32_t *)0, (uint32_t*)0, d_RecordHeadersSCAN, context);

		// retrieve last value, the one past the end of the actual values.
		uint32_t * h_recordstablecount = (uint32_t *)malloc(sizeof(uint32_t));

		// want the record at the end of the scan.  the pointer math adds CsvFileLength as uint32_t so equals a CsvFileLength * 4 bytes adjustment
		checkCudaErrors(hipMemcpy((void*)h_recordstablecount, (void*)(d_RecordHeadersSCAN + chunkbytes), 4, hipMemcpyDeviceToHost));
		uint32_t recordstablecount = *h_recordstablecount;
		free(h_recordstablecount);

		for (int idx = 0; idx < numtotalcolumns; idx++)
		{
			// only malloc when there is a positive width.
			if (G_h_fieldbytewidths[idx] > 0)
			{
				int bytessize = G_h_fieldbytewidths[idx] * recordstablecount;

				checkCudaErrors(hipMalloc((void **)&h_d_fieldptrs[idx], bytessize));

				checkCudaErrors(hipHostMalloc((void **)&h_fieldptrs[idx], bytessize));
				printf("HOST MALLOC'ED ARR.ELEM %d FOR CHUNK %d: %llx.\r\n", idx, chunknum, (int64_t)h_fieldptrs[idx]);
			}
			else h_d_fieldptrs[idx] = 0;
		}

		bool cpy = FixDestFields((const void*)h_fieldUTF8charsizes, (const void*)G_h_fieldbytewidths, (size_t)(numtotalcolumns * sizeof(uint16_t)), (const void*)h_d_fieldptrs, (size_t)(numtotalcolumns * sizeof(unsigned char *)));

		printf("Starting Scan Column Headers.\n");
		Scan<MgpuScanTypeExc>(d_ColumnHeaders_printing, chunkbytesplus1,
			(uint32_t)0, mgpu::plus<uint32_t>(), (uint32_t *)0, (uint32_t*)0, d_ColumnHeadersSCAN, context);

		// retrieve last value, the one past the end of the actual values.
		// want the column at the end of the scan.  the pointer math adds CsvFileLength as uint32_t so equals a CsvFileLength * 4 bytes adjustment
		uint32_t columnstablecount = 0;
		checkCudaErrors(hipMemcpy((void*)&columnstablecount, (void*)(d_ColumnHeadersSCAN + chunkbytes), 4, hipMemcpyDeviceToHost));

		launch_RecordsColumnsChars_StreamCompact(d_RecordHeaders_printing, d_ColumnHeaders_printing, d_UTF8Headers, d_RecordHeadersSCAN, d_ColumnHeadersSCAN, d_UTF8HeadersSCAN,
			d_RecordsTable, d_ColumnsTable, d_UTF8CharsTable, d_RecordsToColumnsTable, d_ColumnsToUTF8charsTable, chunkbytesplus1);

		launch_GetRecLengthsAndColCountErrorsSMEM2(d_RecordsTable, d_RecordsToColumnsTable, d_RecordLengths, d_ColumnCountErrors, d_ColumnCountsPerRecordTable, recordstablecount, numtotalcolumns);

		launch_GetColumnNumsInRecords(d_RecordHeadersSCAN, d_ColumnHeadersSCAN, d_RecordsToColumnsTable, d_ColumnNumInRecord, chunkbytes);

		launch_GetCharNumsInColumns(d_ColumnHeadersSCAN, d_UTF8HeadersSCAN, d_ColumnsToUTF8charsTable, d_CharNumInColumn, chunkbytes);

		// Get the char count in column overflow errors.
		// VERSION THAT MERGES ERRORS WITH COL COUNT ERRORS.
		launch_GetColumnCharCountOverflowErrorsMERGE(d_RecordHeadersSCAN, d_UTF8Headers, d_ColumnNumInRecord, d_CharNumInColumn, d_ColumnCountErrors, chunkbytes, numtotalcolumns);

		printf("Starting Scan Errors Headers.\n");

		Scan<MgpuScanTypeExc>(d_ColumnCountErrors, recordstablecount + 1,
			(uint32_t)0, mgpu::plus<uint32_t>(), (uint32_t *)0, (uint32_t*)0, d_ColumnCountErrorsSCAN, context);

		// retrieve last value, the one past the end of the actual values.
		// want the column at the end of the scan.  the pointer math makes the 4 bytes adjustment by type of pointer.
		uint32_t columncounterrorscount;
		checkCudaErrors(hipMemcpy((void*)&columncounterrorscount, (void*)(d_ColumnCountErrorsSCAN + recordstablecount), 4, hipMemcpyDeviceToHost));

		checkCudaErrors(hipHostMalloc((void**)&h_ColumnCountErrorsTable_a, (columncounterrorscount * sizeof(uint32_t)) + 128));

		launch_SimpleStreamCompact(d_ColumnCountErrors, d_ColumnCountErrorsSCAN, d_ColumnCountErrorsTable, recordstablecount + 1);

		checkCudaErrors(hipMemcpy((void*)h_ColumnCountErrorsTable_a, (void*)d_ColumnCountErrorsTable, columncounterrorscount * sizeof(uint32_t), hipMemcpyDeviceToHost));

		printf("Write CSV record.\r\n");

		for (int idx = 0; idx < numtotalcolumns; idx++)
		{
			// only memset when there is a positive width.
			if (G_h_fieldbytewidths[idx] > 0)
			{
				//cuda malloc for char size <<plus 1 (for null term)>> time 3 for utf8 explansion time number of records in CSV file.
				int bytessize = G_h_fieldbytewidths[idx] /* * (int)charmult*/ * recordstablecount;

				checkCudaErrors(hipMemset((void*)h_d_fieldptrs[idx], 0, bytessize));
			}
		}

		launch_WriteCSVRecord2((uint8_t *)d_CsvBuffer_printing, d_RecordsTable,
			d_ColumnCountErrors, d_ColumnCountErrorsSCAN, d_ColumnsTable, d_RecordsToColumnsTable,
			/* d_fieldcharsizes, d_fieldptrs, */
			numtotalcolumns, recordstablecount/*, charmult*/);

		for (int idx = 0; idx < numtotalcolumns; idx++)
		{
			// only memcpy/free when there is a positive width.
			if (G_h_fieldbytewidths[idx] > 0)
			{
				//cuda malloc for char size <<plus 1 (for null term)>> time 3 for utf8 explansion time number of records in CSV file.
				int bytessize = G_h_fieldbytewidths[idx] /* * (int)charmult*/ * recordstablecount;

				checkCudaErrors(hipMemcpy((void*)h_fieldptrs[idx], (void*)h_d_fieldptrs[idx], bytessize, hipMemcpyDeviceToHost));
				Check_cuda_Free((void **)&h_d_fieldptrs[idx]);  // free up the GPU buf now.
			}
		}

		// now manage the link lists of memblocks on host side.
		// for the first chunk, create the link lists.
		if (chunknum == 0)
		{
			// create new bases.

			// allocate for saved field count only, however, reference from original CSV column count.
			h_llbases = 0;
			checkCudaErrors(hipHostMalloc((void **)&h_llbases, savedfieldcount * sizeof(memlinklist_base)));

			int sidx = 0;
			for (int idx = 0; idx < numtotalcolumns; idx++)
			{
				// copy to the link list only saved columns.
				if (G_h_fieldbytewidths[idx] > 0)
				{
					// first create a member record and assign current block to it.
					memlinklist_member * h_ll_memb = 0;
					checkCudaErrors(hipHostMalloc((void **)&h_ll_memb, sizeof(memlinklist_member)));

					(*h_ll_memb).h_cur_block_ptr = (void*)h_fieldptrs[idx];
					(*h_ll_memb).cur_block_validrecordcount = recordstablecount - columncounterrorscount;
					(*h_ll_memb).cur_block_errorcount = columncounterrorscount;
					(*h_ll_memb).next = NULL;

					h_llbases[sidx].recordwidth = G_h_fieldbytewidths[idx];
					h_llbases[sidx].first = h_ll_memb;
					h_llbases[sidx].totalvalidrecords = recordstablecount - columncounterrorscount;
					h_llbases[sidx].totalerrors = columncounterrorscount;

					sidx++;
				}
			}
		}
		// after 1st chunk add to linked lists.
		else
		{
			int sidx = 0;
			for (int idx = 0; idx < numtotalcolumns; idx++)
			{
				// copy to the link list only saved columns.
				if (G_h_fieldbytewidths[idx] > 0)
				{
					// make a new member for the current array block.
					memlinklist_member * h_ll_memb = 0;
					checkCudaErrors(hipHostMalloc((void **)&h_ll_memb, sizeof(memlinklist_member)));

					(*h_ll_memb).h_cur_block_ptr = (void*)h_fieldptrs[idx];
					(*h_ll_memb).cur_block_validrecordcount = recordstablecount - columncounterrorscount;
					(*h_ll_memb).cur_block_errorcount = columncounterrorscount;
					(*h_ll_memb).next = NULL;

					// now find next open slot in link list to place it.
					memlinklist_member * start = h_llbases[sidx].first;
					while ((*start).next != NULL)
					{
						start = (*start).next;
					}

					(*start).next = h_ll_memb;  // the open "next" pointer is set to new member.

					h_llbases[sidx].totalvalidrecords += (recordstablecount - columncounterrorscount);  // bump total records count with this block.
					h_llbases[sidx].totalerrors += columncounterrorscount;  // bump total errors count with this block.

					sidx++;
				}
			}
		}

		// free the memory that is re-used by subsequent chunks.
		validrecordscount = recordstablecount - columncounterrorscount;  // save global for debug purposes below.

		Check_cuda_FreeHost((void **)&h_ColumnCountErrorsTable_a);
	}  // end of chunks loop

	// clean up
	DeinitializeGPUElements_REUSABLES();

	// CPU versions for debugging.
	DeinitializeCPUElements_REUSABLES(SufficientBytes);

	// now build CPU or GPU arrays per the linked list based on GPUResidentFlag.

	// RESET FIELD COUNT NOW TO SAVED FIELD COUNT.  NO MORE NEED TO ACCOUNT FOR UNUSED CSV COLUMNS.
	// First reset the byte widths to their new positions.
	int newfi = 0;
	for (int fi = 0; fi < numtotalcolumns; fi++)
	{
		if (G_h_fieldbytewidths[fi] == 0) continue;  // skip a 0 byte width.
		G_h_fieldbytewidths[newfi] = G_h_fieldbytewidths[fi];  // this should be fine as only copying to same or down.
		// Also set the final offset alignment for the return array
		dataColumnOffsets[newfi] = G_h_fieldbytewidths[fi];
		newfi++;  // bump new index once written.
	}

	// note can use 1 of the bases since all should have the same total count.
	G_totalvalidCSVrecordscount = h_llbases[0].totalvalidrecords;

	for (int idx = 0; idx < savedfieldcount; idx++)
	{
		int bytesperrecord = h_llbases[idx].recordwidth;

		//cuda malloc for the total valid bytes.
		uint64_t totalbytessize = bytesperrecord * G_totalvalidCSVrecordscount;
		if(GPUResidentFlag == true)
			checkCudaErrors(hipMalloc((void **)&dataColumnPtrs[idx], totalbytessize));
		else
			checkCudaErrors(hipHostMalloc((void **)&dataColumnPtrs[idx], totalbytessize));

		// now copy over each of the chunk arrays from the host.
		int chunkcounter = 0;
		int cumbytescopied = 0;

		// read in the first member.
		memlinklist_member * start = h_llbases[idx].first;
		int curcount = (*start).cur_block_validrecordcount;
		void * h_curblock = (void*)(*start).h_cur_block_ptr;
		if (GPUResidentFlag == true)
			checkCudaErrors(hipMemcpy((void*)dataColumnPtrs[idx], h_curblock, (size_t)(bytesperrecord*curcount), hipMemcpyHostToDevice));
		else
			memcpy((void*)dataColumnPtrs[idx], h_curblock, (size_t)(bytesperrecord*curcount));

		// can free host mem block now.
		printf("HOST FREE ARR.ELEM %d COMBINING CHUNKS @ CHUNK %d: %llx.\r\n", idx, chunkcounter, (int64_t)h_curblock);
		Check_cuda_FreeHost((void **)&h_curblock);

		cumbytescopied = (bytesperrecord*curcount);

		while ((*start).next != NULL)
		{
			chunkcounter++;

			start = (*start).next;
			curcount = (*start).cur_block_validrecordcount;
			h_curblock = (void*)(*start).h_cur_block_ptr;
			if (GPUResidentFlag == true)
				checkCudaErrors(hipMemcpy((void*)(dataColumnPtrs[idx] + cumbytescopied), h_curblock, (size_t)(bytesperrecord*curcount), hipMemcpyHostToDevice));
			else
				memcpy((void*)(dataColumnPtrs[idx] + cumbytescopied), h_curblock, (size_t)(bytesperrecord*curcount));

			// can free host mem block now.
			printf("HOST FREE ARR.ELEM %d COMBINING CHUNKS @ CHUNK %d: %llx.\r\n", idx, chunkcounter, (int64_t)h_curblock);
			Check_cuda_FreeHost((void **)&h_curblock);

			cumbytescopied += (bytesperrecord*curcount);
		}
	}

	Check_cuda_FreeHost((void **)&h_llbases);  // After freeing all sets of members, free the bases.

	// free up the array of pointers.
	printf("HOST FREE ARRAY CARRIAGE (host): %llx.\r\n", (int64_t)h_fieldptrs);
	Check_cuda_FreeHost((void **)&h_fieldptrs);
	printf("HOST FREE UTF8 CHAR SIZES (host): %llx.\r\n", (int64_t)h_fieldUTF8charsizes);
	Check_cuda_FreeHost((void **)&h_fieldUTF8charsizes);

	return G_totalvalidCSVrecordscount;
}  // end importer_varcols()




extern "C" uint64_t CSVImporterMain(char * filename, char delimiter, uint16_t numTotalColumns, uint16_t numDefinedColumns, int16_t * ColumnCharWidths, unsigned char ** dataColumnPtrs, unsigned int * dataColumnOffsets, int64_t seekafterhdr, uint8_t charmultiplier, bool GPUResidentFlag)
{
	// Get the Cuda device with the most GFLOPS for this operation
	ContextPtr context = CreateCudaDevice(gpuGetMaxGflopsDeviceId());

	checkCudaErrors(hipDeviceSetCacheConfig(hipFuncCachePreferL1));

	uint64_t vcret; // return val for varcols calls.
	vcret = importer_varcols(*context, filename, ColumnCharWidths, numDefinedColumns, numTotalColumns, delimiter, GPUResidentFlag, dataColumnPtrs, dataColumnOffsets, seekafterhdr, charmultiplier);
	
	return vcret;
}


